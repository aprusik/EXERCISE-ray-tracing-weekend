#include "hip/hip_runtime.h"
#include "ray.h"

#include <iostream>
#include <fstream>

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ float hit_sphere(const vec3& center, float radius, const ray& r) {
    vec3 oc = r.origin() - center;
    float a = dot(r.direction(), r.direction());
    float b = 2.0f * dot(oc, r.direction());
    float c = dot(oc, oc) - radius*radius;
    float discriminant = b*b -4*a*c;
    
    if (discriminant < 0) { // no hit
        return -1.0;
    }
    else {                  // hit
        return (-b - sqrt(discriminant) ) / (2.0f*a);
    }
}

__device__ vec3 color(const ray& r) {
    // get location of (possible) sphere hit
    float t = hit_sphere(vec3(0,0,-1), 0.5, r);
    if (t > 0.0) {  // is a hit
        // calculate surface normal at hit location
        vec3 N = unit_vector(r.point_at_parameter(t) - vec3(0,0,-1));
        return 0.5f*vec3(N.x()+1, N.y()+1, N.z()+1);
    }
    // no hit, so render background
    vec3 unit_direction = unit_vector(r.direction());
    t = 0.5f*(unit_direction.y() + 1.0);
    return (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
}

__global__ void render(vec3 *fb, int max_x, int max_y,
                       vec3 lower_left_corner, vec3 horizontal, vec3 vertical, vec3 origin) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || j >= (max_y)) return;
    int pixel_index = j*max_x + i;
    float u = float(i) / float(max_x);
    float v = float(j) / float(max_y);
    ray r(origin, lower_left_corner + u*horizontal + v*vertical);
    fb[pixel_index] = color(r);
}

int main() {
    int nx = 1200;
    int ny = 600;

    int tx = 16;
    int ty = 16;

    vec3 lower_left_corner(-2.0, -1.0, -1.0);
    vec3 horizontal(4.0, 0.0, 0.0);
    vec3 vertical(0.0, 2.0, 0.0);
    vec3 origin(0.0, 0.0, 0.0);

    int num_pixels = nx*ny;
    size_t fb_size = num_pixels*sizeof(vec3);

    // allocate FB
    vec3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // Render our buffer
    dim3 blocks(nx/tx+1, ny/ty+1);
    dim3 threads(tx,ty);
    render<<<blocks, threads>>>(fb, nx, ny, lower_left_corner, horizontal, vertical, origin);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // open file and add header
    std::ofstream outfile;
    outfile.open ("render.ppm");
    outfile << "P3\n" << nx << " " << ny << "\n255\n";

    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*nx + i;
            int ir = int(255.99*fb[pixel_index].r());
            int ig = int(255.99*fb[pixel_index].g());
            int ib = int(255.99*fb[pixel_index].b());
            outfile << ir << " " << ig << " " << ib << "\n";
        }
    }
    checkCudaErrors(hipFree(fb));
}